#include "hip/hip_runtime.h"
// Cuda implementation of capped_distance

#include <cstdio>
#include <vector>

#include "capped_distance.h"

#define checkReturn(ans) gpuAssert((ans), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void distanceMatrix(const double* d1, size_t d1_size, const double* d2, size_t d2_size, double* result) {
    size_t d1_idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t d2_idx = threadIdx.y + blockIdx.y * blockDim.y;

    if (d1_idx >= d1_size || d2_idx  >= d2_size) {
        return;
    }

    // Need to account for 3 floats in an element
    double dx = d1[3 * d1_idx] - d2[3 * d2_idx];
    double dy = d1[3 * d1_idx + 1] - d2[3 * d2_idx + 1];
    double dz = d1[3 * d1_idx + 2] - d2[3 * d2_idx + 2];


    size_t result_idx = d1_idx * d2_size + d2_idx;
    result[result_idx] = sqrt(dx * dx + dy * dy + dz * dz);
}

Distances CappedDistanceCuda(const std::vector<DVec>& d1, const std::vector<DVec>& d2, double cap) {
    Distances output;
    if (d1.empty() || d2.empty()) {
        return output;
    }

    // Reinterpret as contiguous doubles and compute number of bytes
    const double* d1_ptr = &d1[0][0];
    const double* d2_ptr = &d2[0][0];
    size_t d1_size = d1.size() * 3 * sizeof(double);
    size_t d2_size = d2.size() * 3 * sizeof(double);

    // Allocate and copy over to the GPU
    double* d1_dev_ptr = nullptr;
    double* d2_dev_ptr = nullptr;
    double* multiplication_matrix_dev_ptr = nullptr;
    size_t multiplication_matrix_size = d1.size() * d2.size() * sizeof(double);

    checkReturn(hipMalloc(&d1_dev_ptr, d1_size));
    checkReturn(hipMalloc(&d2_dev_ptr, d2_size));
    checkReturn(hipMalloc(&multiplication_matrix_dev_ptr, multiplication_matrix_size));

    checkReturn(hipMemcpy(d1_dev_ptr, d1_ptr, d1_size, hipMemcpyHostToDevice));
    checkReturn(hipMemcpy(d2_dev_ptr, d2_ptr, d2_size, hipMemcpyHostToDevice));

    // debuggy thign, remove
    hipMemset(multiplication_matrix_dev_ptr, -2.0, multiplication_matrix_size);



    // Execute kernel
    constexpr size_t blockStride = 16;
    const size_t numBlocks_d1 = d1.size() % blockStride == 0 ? d1.size() / blockStride : d1.size() / blockStride + 1;
    const size_t numBlocks_d2 = d2.size() % blockStride == 0 ? d2.size() / blockStride : d2.size() / blockStride + 1;
    const dim3 gridDimensions(numBlocks_d1, numBlocks_d2);
    const dim3 blockDimensions(16, 16);


    distanceMatrix<<<gridDimensions,blockDimensions>>>(d1_dev_ptr, d1.size(), d2_dev_ptr, d2.size(), multiplication_matrix_dev_ptr);
    // Copy results back over
    std::vector<double> results(d1.size() * d2.size(), -1);
    checkReturn(hipMemcpy(results.data(), multiplication_matrix_dev_ptr, multiplication_matrix_size, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < d1.size(); i++) {
        for (size_t j = 0; j < d2.size(); j++) {
            size_t index = i * d2.size() + j;
            if (double d = results[index]; d < cap) {
                output.idx1.push_back(i);
                output.idx2.push_back(j);
                output.distances.push_back(d);
            }
        }
    }

    checkReturn(hipFree(d1_dev_ptr));
    checkReturn(hipFree(d2_dev_ptr));
    checkReturn(hipFree(multiplication_matrix_dev_ptr));
    return output;
}
